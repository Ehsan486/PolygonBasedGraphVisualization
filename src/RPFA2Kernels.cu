#include "hip/hip_runtime.h"

#include <cstdio>
#include <stdio.h>
#include "RPFA2Kernels.cuh"
#include "RPBHFA2LaunchParameters.cuh"
#include <cfloat>
/// Some variables for FA2 related to `speed'
static __device__ float k_s_maxd = 10.0;
static __device__ float global_speedd = 1.0;
static __device__ float speed_efficiencyd = 1.0;
static __device__ float jitter_toleranced = 1.0;
static __device__ unsigned int blkcntd_speed_kernel = 0;
__global__
__launch_bounds__(THREADS6, FACTOR6) void GravityKernel(int nbodiesd, const float k_g, const bool strong_gravity,
                                                        volatile float *__restrict body_massd,
                                                        volatile float2 *__restrict body_posd,
                                                        volatile float *__restrict fxd, volatile float *__restrict fyd,
                                                        float Cx, float Cy, float *d_points, int *d_externa_polygonX, float *d_externa_polygonX_x, float *d_externa_polygonX_y, int max_degree, const float k_r, int num_points)
{
    register int i, inc;

    // iterate over all bodies assigned to thread
    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < nbodiesd; i += inc)
    {
        if (i >= nbodiesd)
            continue;

        const float px = body_posd[i].x;
        const float py = body_posd[i].y;
        const float epsilon = 0.0000001;
        // Distance from centroid to node (d_n)
        const float dx_c = px - Cx;
        const float dy_c = py - Cy;
        const float d_n = sqrtf(dx_c * dx_c + dy_c * dy_c);

        // Find d_c: the distance from the centroid to the polygon edge along the line passing through the node
        float d_c = FLT_MAX; // Initialize with a large number

        for (int j = 0; j < num_points; ++j)
        {
            float x1 = d_points[2 * j];
            float y1 = d_points[2 * j + 1];
            float x2 = d_points[2 * ((j + 1) % num_points)];
            float y2 = d_points[2 * ((j + 1) % num_points) + 1];

            // Vector from centroid to node (line direction)
            float dx_cn = dx_c;
            float dy_cn = dy_c;

            // Vector from edge start to end
            float edge_dx = x2 - x1;
            float edge_dy = y2 - y1;

            // Find intersection along the line from centroid to node with the polygon edge
            float denom = dx_cn * edge_dy - dy_cn * edge_dx;

            if (fabsf(denom) > epsilon)
            { // Ensure no division by zero (parallel check)
                float t = ((Cx - x1) * edge_dy - (Cy - y1) * edge_dx) / denom;

                // Intersection point along the centroid-node line (d_c)
                float intersect_x = Cx + t * dx_cn;
                float intersect_y = Cy + t * dy_cn;

                // Check if the intersection point is within the edge bounds
                float edge_t = ((intersect_x - x1) * edge_dx + (intersect_y - y1) * edge_dy) / ((edge_dx * edge_dx + edge_dy * edge_dy)+epsilon);
                if (edge_t >= 0.0f && edge_t <= 1.0f)
                {
                    float dist_to_edge = sqrtf((Cx - intersect_x) * (Cx - intersect_x) +
                                               (Cy - intersect_y) * (Cy - intersect_y));
                    d_c = fminf(d_c, dist_to_edge); // Keep the minimum distance to any edge
                }
            }
        }

        // Calculate d_n' as the difference: d_n' = d_c - d_n
        float d_n_prime = fabsf(d_c - d_n);

        // Compute gravitational force using the modified equation
        float f_g = 0.0f;

        if (d_n > d_c)
            f_g = k_g * body_massd[i] / (d_n - d_c);
        else if (d_n > (d_c * .7) && d_n != d_c)
            f_g = k_g * body_massd[i] / (d_c*.5);
                else if (d_n != 0)
                    f_g = k_g * body_massd[i] / (d_n);

        // f_g = k_g * body_massd[i]  / (d_n);

        // Apply gravitational force in the direction toward the centroid
        fxd[i] += (-dx_c * f_g);
        fyd[i] += (-dy_c * f_g);
    }
}
__global__ __launch_bounds__(THREADS6, FACTOR6) void exGravityKernel2(int nbodiesd, const float k_g, const bool strong_gravity,
                                                                      volatile float *__restrict body_massd,
                                                                      volatile float2 *__restrict body_posd,
                                                                      volatile float *__restrict fxd, volatile float *__restrict fyd,
                                                                      float Cx, float Cy, float *d_points, int *d_externa_polygonX,
                                                                      float *d_externa_polygonX_x, float *d_externa_polygonX_y,
                                                                      int max_degree, int num_polygons, int num_points, float *d_effective_d)
{
    register int i, inc;
    inc = blockDim.x * gridDim.x;
    const float epsilon = 1e-6f;

    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < nbodiesd; i += inc)
    {
        if (i >= nbodiesd)
            continue;

        const float px = body_posd[i].x;
        const float py = body_posd[i].y;
        float total_fx = 0.0f;
        float total_fy = 0.0f;
        float Cdx = px - Cx;
        float Cdy = py - Cy;
        // Check for external score and apply external forces if present
        for (int j = 0; j < 500; j++)
        {

            if (d_externa_polygonX[i * 500 + j] > 0)
            {
                float ex = d_externa_polygonX_x[j];
                float ey = d_externa_polygonX_y[j];
                float edx = px - ex;
                float edy = py - ey;
                if (sqrt(edx * edx + edy * edy)  > 0)
                {
                    float f_g = k_g *body_massd[i]*2  / ((sqrtf(edx * edx + edy * edy) *(d_externa_polygonX[i * 500 + j]) ));
                    if (body_massd[i] <= 1)
                        f_g = f_g * sqrtf(max_degree);
                    total_fx += -edx * f_g;
                    total_fy += -edy * f_g;
                }
            }
        }

        // Apply accumulated forces
        fxd[i] += total_fx;
        fyd[i] += total_fy;
    }
}
__global__ __launch_bounds__(THREADS6, FACTOR6) void cornerKernel(int nbodiesd, const float k_g, const bool strong_gravity,
                                                                  volatile float *__restrict body_massd,
                                                                  volatile float2 *__restrict body_posd,
                                                                  volatile float *__restrict fxd, volatile float *__restrict fyd,
                                                                  float Cx, float Cy, float *d_points, int *d_externa_polygonX,
                                                                  float *d_externa_polygonX_x, float *d_externa_polygonX_y,
                                                                  int max_degree, int num_polygons, int num_points, float *d_effective_d, float max_distance_corner)
{
    register int i, inc;
    inc = blockDim.x * gridDim.x;
    const float epsilon = 1e-6f;

    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < nbodiesd; i += inc)
    {
        if (i >= nbodiesd)
            continue;

        const float px = body_posd[i].x;
        const float py = body_posd[i].y;
        float total_fx = 0.0f;
        float total_fy = 0.0f;
        float Cdx = px - Cx;
        float Cdy = py - Cy;
        bool has_external_score = false;
        // Check for external score and apply external forces if present

        // Apply additional gravity forces toward sorted corner points
        // if(!has_external_score)
        for (int j = 0; j < num_points; j++)
        {

            int corner_idx = j; // centroid_indices[j];
            float effective_d = d_effective_d[j];
            float corner_x = d_points[2 * corner_idx];
            float corner_y = d_points[2 * corner_idx + 1];
            float dx = px - corner_x;
            float dy = py - corner_y;
            float dist = sqrtf((dx * dx + dy * dy));
            if (dist > effective_d&&dist>0)
            {
                float f_g_extra = (k_g*body_massd[i]) / (dist);
                total_fx += -dx * f_g_extra;
                total_fy += -dy * f_g_extra;
            }
        }

        // Apply accumulated forces
        fxd[i] += total_fx;
        fyd[i] += total_fy;
    }
}
__global__
__launch_bounds__(THREADS6, FACTOR6) void AttractiveForceKernel2(int nedgesd, int nbodiesd, volatile float *__restrict body_massd, float maximum_area,
                                                                 volatile float2 *__restrict body_posd,
                                                                 volatile float *__restrict fxd, volatile float *__restrict fyd,
                                                                 volatile int *__restrict sourcesd, volatile int *__restrict targetsd, float scale, int *d_externa_polygonX, float *d_externa_polygonX_x, float *d_externa_polygonX_y)
{
    int i, inc;
    inc = blockDim.x * gridDim.x;

    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < nedgesd; i += inc)
    {
        if (i >= nedgesd)
            continue;
        if(nbodiesd>0){
        const float min_distance = sqrt((maximum_area) / nbodiesd);
        int source = sourcesd[i];
        int target = targetsd[i];

        // dx and dy are distance between source and target nodes.
        float dx = body_posd[target].x - body_posd[source].x;
        float dy = body_posd[target].y - body_posd[source].y;

        // Current distance between the nodes
        float distance = sqrtf(dx * dx + dy * dy);

        // Desired distance after scaling
        float desired_distance = distance + (min_distance );

        // Adjust positions proportionally to achieve the desired distance
        if (distance > 0)
        { // Avoid division by zero
            float adjustment_factor = (desired_distance - distance) / distance;
            float force_x = dx * adjustment_factor;
            float force_y = dy * adjustment_factor;

            atomicAdd((float *)fxd + source, -force_x);
            atomicAdd((float *)fyd + source, -force_y);
            atomicAdd((float *)fxd + target, force_x);
            atomicAdd((float *)fyd + target, force_y);
        }
    }}
}
__global__
__launch_bounds__(THREADS6, FACTOR6) void AttractiveForceKernel(int nedgesd, int nbodiesd, volatile float *__restrict body_massd, float maximum_area,
                                                                volatile float2 *__restrict body_posd,
                                                                volatile float *__restrict fxd, volatile float *__restrict fyd,
                                                                volatile int *__restrict sourcesd, volatile int *__restrict targetsd, float scale)
{
    register int i, inc, source, target;
    // iterate over all edges assigned to thread
    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < nedgesd; i += inc)
    {
        if (i >= nedgesd)
            continue;

        source = sourcesd[i];
        target = targetsd[i];

        // dx and dy are distance to between the neighbors.
        const float dx = body_posd[target].x - body_posd[source].x;
        const float dy = body_posd[target].y - body_posd[source].y;

        // Calculate the Euclidean distance between the source and target nodes

        const float fsx = dx;
        const float fsy = dy;
        const float ftx = -dx;
        const float fty = -dy;

        atomicAdd((float *)fxd + source, fsx);
        atomicAdd((float *)fyd + source, fsy);
        atomicAdd((float *)fxd + target, ftx);
        atomicAdd((float *)fyd + target, fty);
    }
}

__global__
__launch_bounds__(THREADS1, FACTOR1) void SpeedKernel(int nbodiesd,
                                                      volatile float *__restrict fxd, volatile float *__restrict fyd,
                                                      volatile float *__restrict fx_prevd, volatile float *__restrict fy_prevd,
                                                      volatile float *__restrict body_massd, volatile float *__restrict swgd, volatile float *__restrict etrad)
{
    register int i, j, k, inc;
    register float swg_thread, swg_body, etra_thread, etra_body, dx, dy, mass;
    // setra: effective_traction (in shared mem.)
    // sswg: swing per node (in shared mem.)
    __shared__ volatile float sswg[THREADS1], setra[THREADS1];

    // initialize with valid data (in case #bodies < #threads)
    swg_thread = 0;
    etra_thread = 0;

    // scan all bodies
    i = threadIdx.x;
    inc = THREADS1 * gridDim.x;

    for (j = i + blockIdx.x * THREADS1; j < nbodiesd; j += inc)
    {
        if (i >= nbodiesd)
            continue;
        mass = body_massd[j];

        dx = fxd[j] - fx_prevd[j];
        dy = fyd[j] - fy_prevd[j];
        swg_body = sqrtf(dx * dx + dy * dy);
        swg_thread += mass * swg_body;

        dx = fxd[j] + fx_prevd[j];
        dy = fyd[j] + fy_prevd[j];
        etra_body = sqrtf(dx * dx + dy * dy) / 2.0;
        etra_thread += mass * etra_body;
    }

    // reduction in shared memory
    sswg[i] = swg_thread;
    setra[i] = etra_thread;

    for (j = THREADS1 / 2; j > 0; j /= 2)
    {
        __syncthreads();
        if (i < j)
        {
            k = i + j;
            sswg[i] = swg_thread = sswg[i] + sswg[k];
            setra[i] = etra_thread = setra[i] + setra[k];
        }
    }

    // swg_thread and etra_thread are now the total swinging
    // and the total effective traction (accross all threads)

    // write block result to global memory
    if (i == 0)
    {
        k = blockIdx.x;
        swgd[k] = swg_thread;
        etrad[k] = etra_thread;
        __threadfence();

        inc = gridDim.x - 1;
        if (inc == atomicInc(&blkcntd_speed_kernel, inc))
        {
            swg_thread = 0;
            etra_thread = 0;

            for (j = 0; j <= inc; j++)
            {
                swg_thread += swgd[j];
                etra_thread += etrad[j];
            }
            // we need to do some calculations to derive
            // from this the new global speed
            float estimated_optimal_jitter_tollerance = 0.05 * sqrtf(nbodiesd);
            float minJT = sqrtf(estimated_optimal_jitter_tollerance);
            float jt = jitter_toleranced * fmaxf(minJT, fminf(k_s_maxd, estimated_optimal_jitter_tollerance * etra_thread / powf(nbodiesd, 2.0)));
            float min_speed_efficiency = 0.05;

            // `Protect against erratic behavior'
            if (swg_thread / etra_thread > 2.0)
            {
                if (speed_efficiencyd > min_speed_efficiency)
                    speed_efficiencyd *= 0.5;
                jt = fmaxf(jt, jitter_toleranced);
            }

            // `Speed efficiency is how the speed really corrosponds to the swinging vs. convergence tradeoff.'
            // `We adjust it slowly and carefully'
            float targetSpeed = jt * speed_efficiencyd * etra_thread / swg_thread;

            if (swg_thread > jt * etra_thread)
            {
                if (speed_efficiencyd > min_speed_efficiency)
                {
                    speed_efficiencyd *= 0.7;
                }
            }
            else if (global_speedd < 1000)
            {
                speed_efficiencyd *= 1.3;
            }

            // `But the speed shouldn't rise much too quickly, ... would make convergence drop dramatically'.
            float max_rise = 0.5;
            global_speedd += fminf(targetSpeed - global_speedd, max_rise * global_speedd);
        }
    }
}

__global__
__launch_bounds__(THREADS6, FACTOR6) void DisplacementKernel(int nbodiesd,
                                                             volatile float2 *__restrict body_posd,
                                                             volatile float *__restrict fxd, volatile float *__restrict fyd,
                                                             volatile float *__restrict fx_prevd, volatile float *__restrict fy_prevd,
                                                             float *d_points)
{
    register int i, inc;
    register float factor, swg, dx, dy, fx, fy;
    register float global_speed = global_speedd;

    // iterate over all bodies assigned to thread
    inc = blockDim.x * gridDim.x;
    for (i = threadIdx.x + blockIdx.x * blockDim.x; i < nbodiesd; i += inc)
    {
        if (i >= nbodiesd)
            continue;

        fx = fxd[i];
        fy = fyd[i];
        dx = fx - fx_prevd[i];
        dy = fy - fy_prevd[i];
        swg = sqrtf(dx * dx + dy * dy);
        factor = global_speed / (1.0 + sqrtf(global_speed * swg));

        // Update the position
        float new_x = body_posd[i].x + fx * factor;
        float new_y = body_posd[i].y + fy * factor;

        // Only update position if inside the polygon

        body_posd[i].x = new_x;
        body_posd[i].y = new_y;

        // Save previous forces
        fx_prevd[i] = fx;
        fy_prevd[i] = fy;
        fxd[i] = 0.0;
        fyd[i] = 0.0;
    }
}
